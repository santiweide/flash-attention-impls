#include "hip/hip_runtime.h"
/******************************************************************************
 * Minimal Flash Attention Implementation with Cutlass
 * 
 * 这是一个教学性质的Flash Attention实现，保留核心算法但简化了许多细节
 * 
 * 核心思想：
 * 1. 将Q,K,V分块，减少HBM访问
 * 2. 在线更新softmax统计量(max和sum)
 * 3. 在一个kernel中融合所有操作
 * 
 * 简化：
 * - 固定块大小和head维度
 * - 不支持causal mask, dropout等
 * - 仅支持FP16
 * - 简化的内存布局
 ******************************************************************************/

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <cutlass/cutlass.h>
#include <cutlass/numeric_types.h>
#include <cutlass/arch/memory.h>
#include <cmath>
#include <algorithm>

// ==================== 配置参数 ====================

// 如果遇到共享内存不足的问题，可以尝试以下配置：
// 选项1 (当前): 64x64 块，需要 ~72KB 共享内存
// 选项2: 32x32 块，只需要 ~20KB 共享内存
// 选项3: 64x32 块，需要 ~44KB 共享内存

constexpr int kBlockM = 64;      // Q的块大小 (行) - 可以改为32
constexpr int kBlockN = 64;      // K,V的块大小 (列) - 可以改为32  
constexpr int kHeadDim = 64;     // Head维度 (固定)
constexpr int kNThreads = 128;   // 每个block的线程数

// ==================== 工具函数 ====================

// 安全的除法
__device__ __forceinline__ float safe_divide(float a, float b) {
    return b == 0.0f ? 0.0f : a / b;
}

// ==================== Softmax在线更新 ====================

/**
 * 在线softmax更新
 * 
 * 给定旧的max(m_old), sum(l_old)和新的scores(S_new)，
 * 计算更新后的max(m_new), sum(l_new)和新的P = exp(S - m_new)
 * 
 * 算法：
 * m_new = max(m_old, max(S_new))
 * l_new = exp(m_old - m_new) * l_old + sum(exp(S_new - m_new))
 */
struct OnlineSoftmax {
    float m;  // running max
    float l;  // running sum of exp
    
    __device__ OnlineSoftmax() : m(-INFINITY), l(0.0f) {}
    
    // 更新统计量并返回缩放因子
    __device__ void update(const float* scores, float* P, int len) {
        // Step 1: 找到新的最大值
        float m_new = m;
        for (int i = 0; i < len; i++) {
            m_new = fmaxf(m_new, scores[i]);
        }
        
        // Step 2: 计算P = exp(S - m_new)并累加
        float l_new = 0.0f;
        for (int i = 0; i < len; i++) {
            P[i] = expf(scores[i] - m_new);
            l_new += P[i];
        }
        
        // Step 3: 合并旧的统计量
        float scale_old = expf(m - m_new);
        l_new = scale_old * l + l_new;
        
        // 更新
        m = m_new;
        l = l_new;
    }
    
    // 获取缩放因子（用于更新O）
    __device__ float get_scale() const {
        return safe_divide(1.0f, l);
    }
    
    __device__ float get_correction(float m_old) const {
        return expf(m_old - m);
    }
};

// ==================== 共享内存管理 ====================

// 共享内存布局
template<typename T>
struct SharedMemory {
    T* Q;      // [kBlockM, kHeadDim]
    T* K;      // [kBlockN, kHeadDim]
    T* V;      // [kBlockN, kHeadDim]
    float* S;  // [kBlockM, kBlockN] - attention scores
    float* P;  // [kBlockM, kBlockN] - attention weights (after softmax)
    
    __device__ SharedMemory(void* ptr) {
        char* base = reinterpret_cast<char*>(ptr);
        size_t offset = 0;
        
        Q = reinterpret_cast<T*>(base + offset);
        offset += kBlockM * kHeadDim * sizeof(T);
        
        K = reinterpret_cast<T*>(base + offset);
        offset += kBlockN * kHeadDim * sizeof(T);
        
        V = reinterpret_cast<T*>(base + offset);
        offset += kBlockN * kHeadDim * sizeof(T);
        
        S = reinterpret_cast<float*>(base + offset);
        offset += kBlockM * kBlockN * sizeof(float);
        
        P = reinterpret_cast<float*>(base + offset);
        offset += kBlockM * kBlockN * sizeof(float);
    }
    
    static constexpr size_t get_size() {
        size_t base_size = (kBlockM * kHeadDim + kBlockN * kHeadDim * 2) * sizeof(T) +
                          (kBlockM * kBlockN * 2) * sizeof(float);
        // 添加额外的统计量和累加器
        size_t extra_size = (kBlockM * 2) * sizeof(float) +  // m_shared, l_shared
                           (kBlockM * kHeadDim) * sizeof(float);  // O_accum
        return base_size + extra_size;
    }
};

// ==================== GEMM操作 ====================

// 简化的GEMM: C = A @ B^T
// A: [M, K], B: [N, K], C: [M, N]
template<typename T, int M, int N, int K>
__device__ void gemm_nt(const T* A, const T* B, float* C, int lda, int ldb, int ldc) {
    const int tid = threadIdx.x;
    const int num_threads = blockDim.x;
    
    // 每个线程计算C的一部分元素
    for (int idx = tid; idx < M * N; idx += num_threads) {
        int i = idx / N;  // row in C
        int j = idx % N;  // col in C
        
        float sum = 0.0f;
        #pragma unroll 8
        for (int k = 0; k < K; k++) {
            float a_val = float(A[i * lda + k]);
            float b_val = float(B[j * ldb + k]);
            sum += a_val * b_val;
        }
        C[i * ldc + j] = sum;
    }
    __syncthreads();
}

// GEMM: C = A @ B
// A: [M, K], B: [K, N], C: [M, N]
template<typename T, int M, int N, int K>
__device__ void gemm_nn(const float* A, const T* B, T* C, int lda, int ldb, int ldc) {
    const int tid = threadIdx.x;
    const int num_threads = blockDim.x;
    
    for (int idx = tid; idx < M * N; idx += num_threads) {
        int i = idx / N;  // row
        int j = idx % N;  // col
        
        float sum = 0.0f;
        #pragma unroll 8
        for (int k = 0; k < K; k++) {
            float a_val = A[i * lda + k];
            float b_val = float(B[k * ldb + j]);
            sum += a_val * b_val;
        }
        C[i * ldc + j] = T(sum);
    }
    __syncthreads();
}

// ==================== 核心Kernel ====================

/**
 * Flash Attention Kernel (简化版)
 * 
 * 参数：
 *   Q, K, V: [batch, num_heads, seq_len, head_dim]
 *   O: 输出 [batch, num_heads, seq_len, head_dim]
 *   softmax_scale: 缩放因子 (通常是 1/sqrt(head_dim))
 *   batch_size, num_heads, seq_len, head_dim: 形状参数
 * 
 * Grid: (num_blocks_M, batch * num_heads)
 * Block: kNThreads threads
 */
__global__ void flash_attention_kernel(
    const cutlass::half_t* __restrict__ Q,
    const cutlass::half_t* __restrict__ K,
    const cutlass::half_t* __restrict__ V,
    cutlass::half_t* __restrict__ O,
    float softmax_scale,
    int batch_size,
    int num_heads,
    int seq_len,
    int head_dim
) {
    // 断言检查
    assert(head_dim == kHeadDim);
    assert(blockDim.x == kNThreads);
    
    // Block和thread索引
    const int batch_head_idx = blockIdx.y;  // batch * num_heads
    const int q_block_idx = blockIdx.x;     // Q的block索引
    const int tid = threadIdx.x;
    
    const int batch_idx = batch_head_idx / num_heads;
    const int head_idx = batch_head_idx % num_heads;
    
    // 计算Q block的起始位置
    const int q_start = q_block_idx * kBlockM;
    const int q_end = min(q_start + kBlockM, seq_len);
    const int q_size = q_end - q_start;
    
    if (q_size <= 0) return;  // 越界检查
    
    // 计算全局内存偏移
    const int64_t qkv_offset = (batch_idx * num_heads + head_idx) * seq_len * head_dim;
    const cutlass::half_t* Q_ptr = Q + qkv_offset;
    const cutlass::half_t* K_ptr = K + qkv_offset;
    const cutlass::half_t* V_ptr = V + qkv_offset;
    cutlass::half_t* O_ptr = O + qkv_offset;
    
    // 分配共享内存
    extern __shared__ char smem[];
    SharedMemory<cutlass::half_t> shared_mem(smem);
    
    // 从动态共享内存中分配额外的数组
    size_t base_offset = (kBlockM * kHeadDim + kBlockN * kHeadDim * 2) * sizeof(cutlass::half_t) +
                        (kBlockM * kBlockN * 2) * sizeof(float);
    float* m_shared = reinterpret_cast<float*>(smem + base_offset);
    float* l_shared = m_shared + kBlockM;
    float* O_accum = l_shared + kBlockM;
    
    // 加载Q block到共享内存
    for (int idx = tid; idx < q_size * head_dim; idx += blockDim.x) {
        int i = idx / head_dim;
        int j = idx % head_dim;
        shared_mem.Q[i * kHeadDim + j] = Q_ptr[(q_start + i) * head_dim + j];
    }
    __syncthreads();
    
    // 初始化
    for (int i = tid; i < kBlockM; i += blockDim.x) {
        m_shared[i] = -INFINITY;
        l_shared[i] = 0.0f;
    }
    for (int i = tid; i < kBlockM * kHeadDim; i += blockDim.x) {
        O_accum[i] = 0.0f;
    }
    __syncthreads();
    
    // 计算K,V的block数量
    const int num_k_blocks = (seq_len + kBlockN - 1) / kBlockN;
    
    // 遍历所有K,V blocks
    for (int k_block_idx = 0; k_block_idx < num_k_blocks; k_block_idx++) {
        const int k_start = k_block_idx * kBlockN;
        const int k_end = min(k_start + kBlockN, seq_len);
        const int k_size = k_end - k_start;
        
        // 加载K block到共享内存
        for (int idx = tid; idx < k_size * head_dim; idx += blockDim.x) {
            int i = idx / head_dim;
            int j = idx % head_dim;
            shared_mem.K[i * kHeadDim + j] = K_ptr[(k_start + i) * head_dim + j];
        }
        
        // 加载V block到共享内存
        for (int idx = tid; idx < k_size * head_dim; idx += blockDim.x) {
            int i = idx / head_dim;
            int j = idx % head_dim;
            shared_mem.V[i * kHeadDim + j] = V_ptr[(k_start + i) * head_dim + j];
        }
        __syncthreads();
        
        // 计算 S = Q @ K^T (attention scores)
        gemm_nt<cutlass::half_t, kBlockM, kBlockN, kHeadDim>(
            shared_mem.Q, shared_mem.K, shared_mem.S,
            kHeadDim, kHeadDim, kBlockN
        );
        
        // 应用softmax scale
        for (int idx = tid; idx < q_size * k_size; idx += blockDim.x) {
            shared_mem.S[idx] *= softmax_scale;
        }
        __syncthreads();
        
        // 对每一行进行在线softmax更新
        for (int i = 0; i < q_size; i++) {
            if (tid == 0) {
                float* scores = shared_mem.S + i * kBlockN;
                float* P = shared_mem.P + i * kBlockN;
                
                float m_old = m_shared[i];
                float l_old = l_shared[i];
                
                // 找到新的max
                float m_new = m_old;
                for (int j = 0; j < k_size; j++) {
                    m_new = fmaxf(m_new, scores[j]);
                }
                
                // 计算P = exp(S - m_new)
                float l_new = 0.0f;
                for (int j = 0; j < k_size; j++) {
                    P[j] = expf(scores[j] - m_new);
                    l_new += P[j];
                }
                
                // 合并旧的统计量
                float correction = expf(m_old - m_new);
                l_new = correction * l_old + l_new;
                
                // 更新累加器 O = correction * O + P @ V
                for (int d = 0; d < head_dim; d++) {
                    O_accum[i * kHeadDim + d] *= correction;
                }
                
                m_shared[i] = m_new;
                l_shared[i] = l_new;
            }
        }
        __syncthreads();
        
        // 计算 O += P @ V
        for (int i = 0; i < q_size; i++) {
            for (int d = tid; d < head_dim; d += blockDim.x) {
                float sum = 0.0f;
                const float* P_row = shared_mem.P + i * kBlockN;
                for (int j = 0; j < k_size; j++) {
                    sum += P_row[j] * float(shared_mem.V[j * kHeadDim + d]);
                }
                O_accum[i * kHeadDim + d] += sum;
            }
        }
        __syncthreads();
    }
    
    // 最终归一化并写回全局内存
    for (int i = 0; i < q_size; i++) {
        float scale = safe_divide(1.0f, l_shared[i]);
        for (int d = tid; d < head_dim; d += blockDim.x) {
            float val = O_accum[i * kHeadDim + d] * scale;
            O_ptr[(q_start + i) * head_dim + d] = cutlass::half_t(val);
        }
    }
}

// ==================== Host接口 ====================

/**
 * Flash Attention的Host接口
 * 
 * 输入输出都是CUDA device指针
 * 数据格式: [batch, num_heads, seq_len, head_dim]
 */
void flash_attention_forward(
    const cutlass::half_t* Q,
    const cutlass::half_t* K,
    const cutlass::half_t* V,
    cutlass::half_t* O,
    int batch_size,
    int num_heads,
    int seq_len,
    int head_dim,
    hipStream_t stream = 0
) {
    // 检查参数
    assert(head_dim == kHeadDim && "head_dim must equal kHeadDim");
    assert(seq_len > 0 && "seq_len must be positive");
    
    // 计算softmax scale
    float softmax_scale = 1.0f / sqrtf(static_cast<float>(head_dim));
    
    // 计算grid和block大小
    const int num_q_blocks = (seq_len + kBlockM - 1) / kBlockM;
    dim3 grid(num_q_blocks, batch_size * num_heads);
    dim3 block(kNThreads);
    
    // 计算共享内存大小
    size_t smem_size = SharedMemory<cutlass::half_t>::get_size();
    
    // 检查并设置共享内存限制
    // A100默认48KB，我们需要更多
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    
    // 设置最大共享内存配置
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        flash_attention_kernel),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        smem_size
    );
    
    // 打印调试信息（首次调用）
    static bool first_call = true;
    if (first_call) {
        printf("Shared memory info:\n");
        printf("  Required: %zu bytes (%.1f KB)\n", smem_size, smem_size / 1024.0);
        printf("  Available per block: %zu bytes (%.1f KB)\n", 
               prop.sharedMemPerBlock, prop.sharedMemPerBlock / 1024.0);
        printf("  Max per block with opt-in: %zu bytes (%.1f KB)\n",
               prop.sharedMemPerBlockOptin, prop.sharedMemPerBlockOptin / 1024.0);
        
        if (smem_size > prop.sharedMemPerBlock) {
            printf("  ⚠️  WARNING: Required shared memory exceeds default limit!\n");
            printf("  Attempting to use opt-in limit...\n");
        }
        first_call = false;
    }
    
    // 启动kernel
    flash_attention_kernel<<<grid, block, smem_size, stream>>>(
        Q, K, V, O,
        softmax_scale,
        batch_size, num_heads, seq_len, head_dim
    );
    
    // 检查错误
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Flash attention kernel launch failed: %s\n", 
                hipGetErrorString(err));
        fprintf(stderr, "  Grid: (%d, %d), Block: (%d), Shared mem: %zu bytes\n",
                grid.x, grid.y, block.x, smem_size);
    }
}

